#include "hip/hip_runtime.h"
/* Matrix-Vector Multiplication: Y = A*X, A is the matrix, X is the Vector.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil.h>
#include "hipblas.h"

// includes, kernels
#include <mv_kernel.cu>


// declarations, forward
extern "C"

void computeGold(float*, const float*, const float*, unsigned int, unsigned int);
Matrix AllocateDeviceMatrix(const Matrix M);
Matrix AllocateMatrix(int height, int width, int init);
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost);
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice);
int ReadFile(Matrix* M, char* file_name);
void WriteFile(Matrix M, char* file_name);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);
int MVOnDevice(const Matrix A, const Matrix X, Matrix Y);


char choose;
clock_t lapse;
clock_t lapse1;
/*
*******************************************************************************
* Main Program
*******************************************************************************
*/
int main(int argc, char** argv) {

  	Matrix  A;
  	Matrix  X;
  	Matrix  Y;
  	
  	srand(time(NULL));
    printf("----Please choose one method to run the matrix vector multiplication\n");
    printf("----Enter 1 or 2 or 3, then press ENTER button.\n");
    printf("----(1). Global memory\n");
    printf("----(2). shared memory\n");
    printf("----(3). CUBLAS\n");
    printf("----You choose number here: ");
    //choose = getchar();
    for (int iii = 1; iii<=3; iii++)
{   
    if (iii == 1)
      choose = '1';
    else if (iii == 2)
      choose = '2';
    else if (iii == 3)
      choose = '3';

    if (choose == '1' || choose == '2' || choose == '3') 
    	{
        printf("----Iteration number is %d:\n",ITERATIONS);	
      	if(argc != 5 && argc != 4) 
        	{
        		// Allocate and initialize the matrices
                	A  = AllocateMatrix(rand()%1024, rand()%1024, 1);
                        //A  = AllocateMatrix(4096, 4096, 1);
        		X  = AllocateMatrix(A.width, 1, 1);
        		Y  = AllocateMatrix(A.height, 1, 0);
        	}
      	else
        	{
        		// Allocate and read in matrices from disk
        		int* params = NULL; 
        		unsigned int data_read = 0;
        		cutReadFilei(argv[1], &params, &data_read, true);
        		if(data_read != 2){
        			printf("Error reading parameter file\n");
        			cutFree(params);
        			return 1;
        		}
        
        		A  = AllocateMatrix(params[0],params[1], 0);
        		X  = AllocateMatrix(A.width, 1, 0);		
        		Y  = AllocateMatrix(A.height, 1, 0);
        		cutFree(params);
        		(void)ReadFile(&A, argv[2]);
        		(void)ReadFile(&X, argv[3]);
        	}
    
    	// Matrix-Vector Multiplication on the device
    
        printf("Size of MATRIX A is %d by %d.\n",A.height,A.width);
          
        // run the computation on GPU        
        MVOnDevice(A, X, Y);
    
    
        // compute the matrix-vector multiplication on the CPU for comparison
        Matrix reference = AllocateMatrix(Y.height, Y.width, 0);
        // Measure the execution time of CPU implementation	
        lapse = clock();
        for (int i=0;i<ITERATIONS;i++)
        	computeGold(reference.elements, A.elements, X.elements, A.height, A.width);
        lapse = clock()-lapse;
    
        printf("Execution time of CPU implementation is %.6f\n", float(lapse)*1000/(ITERATIONS*CLOCKS_PER_SEC));
      
        // check if the result is equivalent to the expected soluion
        CUTBoolean res = cutComparefe(reference.elements, Y.elements, Y.width * Y.height, 0.001f);
        printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");
        //if (res != 1)
        //printf("----Because it is the paper alg. Please run agian. 90 percent time, it works well.\n");
        //printf("Please...................");
        
        if(argc == 5)
          {
    	       WriteFile(Y, argv[4]);
    	    }
    	  else if(argc == 2)
    	    {
    	       WriteFile(Y, argv[1]);
        	}   
    
    	// Free matrices
        FreeMatrix(&A);
        FreeMatrix(&X);
        FreeMatrix(&Y);
        //return 0;
      }
    else printf("Input wrong.\n");
}
}

/*
*******************************************************************************
* Run Y = A*X
*******************************************************************************
*/
int MVOnDevice(const Matrix A, const Matrix X, Matrix Y)
{
    // Load A and X  to the device
    Matrix Ad = AllocateDeviceMatrix(A);
    CopyToDeviceMatrix(Ad, A);
    Matrix Xd = AllocateDeviceMatrix(X);
    CopyToDeviceMatrix(Xd, X);
    
    // Allocate Y on the device
    Matrix Yd = AllocateDeviceMatrix(Y);
    CopyToDeviceMatrix(Yd, Y);
    // Setup the execution configuration
    //Version 1.0 Global Memory
    dim3 dimGrid((A.height-1)/BLOCK_SIZE+1);
    dim3 dimBlock(BLOCK_SIZE);    
    int m = A.height;
    //int n = A.width;
    int blkNum = (m >> 4) + ((m & 15) ? 1 : 0);
    //int height = blkNum << 4;
    //int width = (n & 255) ? (256*((n >> 8) + 1) ) : n;
    dim3 threads(16, 16);
    dim3 grid(blkNum, 1);
    //Mesaure the time of GPU implementation
    lapse1=clock();	
        for (int i=0;i<ITERATIONS;i++)
          {
              if(choose == '1')	
                {
                    if (i==0) printf("--------You choose global memory method------------\n");
                    MVKernel_gm<<<dimGrid,dimBlock>>>(Ad,Xd,Yd);
                }
              else if (choose == '2')
                {
                    if (i==0) printf("--------You choose shared memory method------------\n");
                    MVKernel_shm<<<dimGrid,dimBlock>>>(Ad,Xd,Yd);
                }
              else if (choose == '3')
                {
                    if (i==0) printf("--------You choose to use CUBLAS method------------\n");
                     hipblasSgemv ('T', Ad.width, Ad.height, 1,
                                   Ad.elements, Ad.width, Xd.elements,
                                   1, 0, Yd.elements, 1);
                    hipblasStatus_t status = cublasGetError();
                    if (status != HIPBLAS_STATUS_SUCCESS) {
                        fprintf (stderr, "!!!! kernel execution error.\n");
                        return EXIT_FAILURE;
                    }    
                }
              else 
                {
                printf("You choose nothing.\n"); break;
                }

          }

    lapse1 = clock()-lapse1;
    printf("\n");
    printf("Execution time of GPU implementation is %.6f\n", float(lapse1)*1000/(ITERATIONS*CLOCKS_PER_SEC));

    // Read Y from the device
    CopyFromDeviceMatrix(Y, Yd);
      
   
    // Free device matrices
    FreeDeviceMatrix(&Ad);
    FreeDeviceMatrix(&Xd);
    FreeDeviceMatrix(&Yd);

}




// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.width * M.height * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a device matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
//  If init == 2, initialize matrix parameters, but do not allocate memory 
Matrix AllocateMatrix(int height, int width, int init)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
    M.elements = NULL;
    
    // don't allocate memory on option 2
    if(init == 2)
		return M;
		
	M.elements = (float*) malloc(size*sizeof(float));

	for(unsigned int i = 0; i < M.height * M.width; i++)
	{
		M.elements[i] = (init == 0) ? (0.0f) : (rand() / (float)RAND_MAX);
		if(rand() % 2)
			M.elements[i] = - M.elements[i];
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.width * Mhost.height * sizeof(float);
    Mdevice.height = Mhost.height;
    Mdevice.width = Mhost.width;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, 
					hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.width * Mdevice.height * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, 
					hipMemcpyDeviceToHost);
}

// Free a device matrix.
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Free a host Matrix
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}

// Read a floating point matrix in from file
int ReadFile(Matrix* M, char* file_name)
{
	unsigned int data_read = M->height * M->width;
	cutReadFilef(file_name, &(M->elements), &data_read, true);
	return data_read;
}

// Write a floating point matrix to file
void WriteFile(Matrix M, char* file_name)
{
    cutWriteFilef(file_name, M.elements, M.width*M.height,
                       0.0001f);
}

