#include "hip/hip_runtime.h"
#ifndef _MV_KERNEL_H_
#define _MV_KERNEL_H_

#include <stdio.h>
#include "mv.h"

// Matrix multiplication kernel thread specification
// 1. Global Memory
__global__ void MVKernel_gm(Matrix A, Matrix X, Matrix Y)
{
	  int bx = blockIdx.x; 
          //int by = blockIdx.y;
	  int tx = threadIdx.x; 
          //int ty = threadIdx.y;
  // Calculate the row index of the Pd element and M
  int Row = bx * BLOCK_SIZE + tx;
  // Calculate the column idenx of Pd and N
  //int Col = bx * BLOCK_SIZE + tx;
  
  float Pvalue = 0;

   
  for (unsigned int k = 0; k < A.width; k++) 
    {
      if(Row < A.height)         
      Pvalue += A.elements[Row*A.width+k] * X.elements[k];
      //else
      //Pvalue += 0;

    }

  __syncthreads();
  
  if(Row < A.height)  		
    Y.elements[Row] = Pvalue;
  __syncthreads();
}


// 2. Shared Memory
__global__ void MVKernel_shm(Matrix A, Matrix X, Matrix Y)
{
   __shared__ float Xds[BLOCK_SIZE];
   
	int bx = blockIdx.x; 
        //int by = blockIdx.y;
	int tx = threadIdx.x; 
        //int ty = threadIdx.y;
  // Calculate the row index
  //int Row = by * BLOCK_SIZE + ty;
  // Calculate the column index
  int Row = bx * BLOCK_SIZE + tx;
  
  float Pvalue = 0;

  for (unsigned int m = 0; m < (A.width-1)/BLOCK_SIZE+1; ++m)
    {
      if(m*BLOCK_SIZE + tx < A.width)
      	Xds[tx] = X.elements[m*BLOCK_SIZE + tx]; 
      else
      	Xds[tx] = 0;   
      __syncthreads();	
      
      for (unsigned int k = 0; k < BLOCK_SIZE; k++) 
             if(Row<A.height && m*BLOCK_SIZE +k<A.width)
    		Pvalue += A.elements[m*BLOCK_SIZE+Row*A.width+k] * Xds[k];    		
    	__syncthreads();
    }		
    
  if(Row < A.height)  
    Y.elements[Row] = Pvalue; 
  __syncthreads();
}

#endif // #ifndef _MV_KERNEL_H_
